#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define SHA256_BLOCK_SIZE 64  // 512 bits
#define SHA256_DIGEST_SIZE 32  // 256 bits

// SHA-256 constants (first 32 bits of the fractional parts of the cube roots of the first 64 primes)
__device__ const uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// Device function for bitwise rotation
__device__ __inline__ uint32_t rotr(uint32_t x, uint32_t n) {
    return (x >> n) | (x << (32 - n));
}

// SHA-256 transformation step
__device__ void sha256_transform(uint32_t state[8], const unsigned char block[SHA256_BLOCK_SIZE]) {
    uint32_t a, b, c, d, e, f, g, h, i, t1, t2, m[64];

    // Prepare message schedule
    for (i = 0; i < 16; ++i) {
        m[i] = (block[i * 4] << 24) | (block[i * 4 + 1] << 16) | (block[i * 4 + 2] << 8) | block[i * 4 + 3];
    }
    for (; i < 64; ++i) {
        m[i] = (rotr(m[i - 2], 17) ^ rotr(m[i - 2], 19) ^ (m[i - 2] >> 10)) + m[i - 7] +
               (rotr(m[i - 15], 7) ^ rotr(m[i - 15], 18) ^ (m[i - 15] >> 3)) + m[i - 16];
    }

    // Initialize working variables to current state
    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    f = state[5];
    g = state[6];
    h = state[7];

    // Compression function main loop
    for (i = 0; i < 64; ++i) {
        t1 = h + (rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25)) + ((e & f) ^ (~e & g)) + k[i] + m[i];
        t2 = (rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22)) + ((a & b) ^ (a & c) ^ (b & c));
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    // Add the compressed chunk to the current hash value
    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

// SHA-256 padding and preprocessing
__device__ void sha256_init(uint32_t state[8]) {
    state[0] = 0x6a09e667;
    state[1] = 0xbb67ae85;
    state[2] = 0x3c6ef372;
    state[3] = 0xa54ff53a;
    state[4] = 0x510e527f;
    state[5] = 0x9b05688c;
    state[6] = 0x1f83d9ab;
    state[7] = 0x5be0cd19;
}

// SHA-256 GPU implementation
__device__ void sha256_gpu(const unsigned char* data, size_t len, unsigned char* hash) {
    uint32_t state[8];
    unsigned char block[SHA256_BLOCK_SIZE];
    size_t i, bit_len = len * 8;

    sha256_init(state);

    // Process each 512-bit chunk
    while (len >= SHA256_BLOCK_SIZE) {
        memcpy(block, data, SHA256_BLOCK_SIZE);
        sha256_transform(state, block);
        data += SHA256_BLOCK_SIZE;
        len -= SHA256_BLOCK_SIZE;
    }

    // Padding
    memcpy(block, data, len);
    block[len] = 0x80;  // Append '1' bit
    if (len < SHA256_BLOCK_SIZE - 8) {
        memset(block + len + 1, 0, SHA256_BLOCK_SIZE - len - 9);
    } else {
        memset(block + len + 1, 0, SHA256_BLOCK_SIZE - len - 1);
        sha256_transform(state, block);
        memset(block, 0, SHA256_BLOCK_SIZE - 8);
    }

    // Append length in bits (big-endian)
    for (i = 0; i < 8; ++i) {
        block[SHA256_BLOCK_SIZE - 1 - i] = bit_len >> (i * 8);
    }
    sha256_transform(state, block);

    // Convert state to hash (big-endian)
    for (i = 0; i < 8; ++i) {
        hash[i * 4] = (state[i] >> 24) & 0xff;
        hash[i * 4 + 1] = (state[i] >> 16) & 0xff;
        hash[i * 4 + 2] = (state[i] >> 8) & 0xff;
        hash[i * 4 + 3] = state[i] & 0xff;
    }
}

// Test kernel to compute SHA-256 on GPU
__global__ void sha256_test_kernel(const unsigned char* data, size_t len, unsigned char* hash) {
    sha256_gpu(data, len, hash);
}

int main() {
    const char* input = "Hello, CUDA SHA-256!";
    unsigned char hash[SHA256_DIGEST_SIZE];

    // Allocate memory on GPU
    unsigned char* d_data;
    unsigned char* d_hash;
    size_t input_len = strlen(input);
    
    hipMalloc((void**)&d_data, input_len);
    hipMalloc((void**)&d_hash, SHA256_DIGEST_SIZE);

    // Copy data to GPU
    hipMemcpy(d_data, input, input_len, hipMemcpyHostToDevice);

    // Launch kernel to compute SHA-256
    sha256_test_kernel<<<1, 1>>>(d_data, input_len, d_hash);

    // Copy hash result back to host
    hipMemcpy(hash, d_hash, SHA256_DIGEST_SIZE, hipMemcpyDeviceToHost);

    // Print the result
    printf("SHA-256 hash: ");
    for (int i = 0; i < SHA256_DIGEST_SIZE; ++i) {
        printf("%02x", hash[i]);
    }
    printf("\n");

    // Free memory
    hipFree(d_data);
    hipFree(d_hash);

    return 0;
}
